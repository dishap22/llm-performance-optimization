#include "hip/hip_runtime.h"
#include <vector>
#include <stdexcept>
#include <numeric>
#include <algorithm> // For std::min

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp> // **FIX**: Include the CUB library for parallel primitives.

#include "spgemm_interface.h"

// Helper macro for CUDA error checking.
#define CUDA_CHECK(err) { \
    hipError_t e = (err); \
    if (e != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(EXIT_FAILURE); \
    } \
}


/**
 * @brief Corrected symbolic kernel using a warp-per-row strategy.
 *
 * Each warp collaborates to find the NNZ for a single row of C.
 * This version uses an efficient warp reduction and correct atomic operations.
 */
__global__ void spgemm_symbolic_warp_per_row(const int num_rows, const int b_cols,
                                             const int* __restrict__ d_A_indptr, const int* __restrict__ d_A_indices,
                                             const int* __restrict__ d_B_indptr, const int* __restrict__ d_B_indices,
                                             int* d_C_row_nnz, int* d_workspace_markers) {
    const int row = blockIdx.x * blockDim.y + threadIdx.y;
    if (row >= num_rows) return;

    const int lane_id = threadIdx.x;
    int* row_markers = d_workspace_markers + static_cast<size_t>(row) * b_cols;
    int nnz = 0; // Each thread counts its successful marks

    const int start_A = d_A_indptr[row];
    const int end_A = d_A_indptr[row + 1];

    // Parallelize the outer loop over A's non-zeros across the warp
    for (int i = start_A + lane_id; i < end_A; i += warpSize) {
        const int col_A = d_A_indices[i];
        const int row_B = col_A;
        const int start_B = d_B_indptr[row_B];
        const int end_B = d_B_indptr[row_B + 1];

        for (int j = start_B; j < end_B; ++j) {
            const int col_B = d_B_indices[j];
            // Atomically mark the column. If we are the first thread to do so, count it.
            // This relies on the workspace slice being 0-initialized.
            if (atomicCAS(row_markers + col_B, 0, 1) == 0) {
                nnz++;
            }
        }
    }

    // Reduce the count across the warp. All threads in a warp add their nnz counts.
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        nnz += __shfl_down_sync(0xFFFFFFFF, nnz, offset);
    }

    // Lane 0 of the warp writes the final, aggregated count for the row.
    if (lane_id == 0) {
        d_C_row_nnz[row] = nnz;
    }
}


/**
 * @brief Corrected numeric kernel using a warp-per-row strategy.
 *
 * This version adds a parallel initialization step for the workspace and
 * uses correct atomic operations for accumulation and compaction.
 */
__global__ void spgemm_numeric_warp_per_row(const int num_rows, const int b_cols,
                                            const int* __restrict__ d_A_indptr, const int* __restrict__ d_A_indices, const float* __restrict__ d_A_data,
                                            const int* __restrict__ d_B_indptr, const int* __restrict__ d_B_indices, const float* __restrict__ d_B_data,
                                            const int* __restrict__ d_C_indptr, int* d_C_indices, float* d_C_data,
                                            float* d_workspace_vals, int* d_workspace_markers) {
    const int row = blockIdx.x * blockDim.y + threadIdx.y;
    if (row >= num_rows) return;

    const int lane_id = threadIdx.x;
    const int warp_id = threadIdx.y;

    float* row_vals = d_workspace_vals + static_cast<size_t>(row) * b_cols;
    int* row_markers = d_workspace_markers + static_cast<size_t>(row) * b_cols;
    const int symbolic_visited_flag = 1;

    // Part 1a: Initialize workspace values to 0 for this row's active columns
    // The warp parallelizes this initialization step.
    for (int j = lane_id; j < b_cols; j += warpSize) {
        if (row_markers[j] == symbolic_visited_flag) {
            row_vals[j] = 0.0f;
        }
    }
    __syncthreads(); // Sync all warps in the block to ensure initialization is complete.

    // Part 1b: Accumulate products in parallel into the workspace.
    const int start_A = d_A_indptr[row];
    const int end_A = d_A_indptr[row + 1];

    for (int i = start_A + lane_id; i < end_A; i += warpSize) {
        const int col_A = d_A_indices[i];
        const float val_A = d_A_data[i];
        const int row_B = col_A;

        const int start_B = d_B_indptr[row_B];
        const int end_B = d_B_indptr[row_B + 1];

        for (int j = start_B; j < end_B; ++j) {
            const int col_B = d_B_indices[j];
            const float val_B = d_B_data[j];
            // Check if this is a valid output column before accumulating
            if (row_markers[col_B] == symbolic_visited_flag) {
                 atomicAdd(&row_vals[col_B], val_A * val_B);
            }
        }
    }
    __syncthreads(); // Sync all warps in the block after accumulation

    // Part 2: Compact the results from the workspace to global memory in parallel.
    const int C_row_start = d_C_indptr[row];
    const int C_row_end = d_C_indptr[row + 1];

    // Use a shared memory counter for this warp's written NNZ
    extern __shared__ int s_C_nnz_written[];
    if (lane_id == 0) {
        s_C_nnz_written[warp_id] = 0;
    }
    __syncthreads();

    // Parallelize the compaction loop across the warp
    for (int j = lane_id; j < b_cols; j += warpSize) {
        if (row_markers[j] == symbolic_visited_flag) {
            int write_pos = atomicAdd(&s_C_nnz_written[warp_id], 1);
            if (C_row_start + write_pos < C_row_end) {
                d_C_indices[C_row_start + write_pos] = j;
                d_C_data[C_row_start + write_pos] = row_vals[j];
            }
        }
    }
}


/**
 * @brief Main host function for GPU-based SpGEMM.
 * This version uses CUB for the parallel scan, which is robust and highly optimized.
 */
void spgemm_gpu(const CSRMatrix& A, const CSRMatrix& B, CSRMatrix& C) {
    if (A.cols != B.rows) {
        throw std::runtime_error("Incompatible matrix dimensions for SpGEMM (A.cols must equal B.rows).");
    }

    // --- 1. Device Memory Allocation & H2D Transfer for Inputs ---
    int *d_A_indptr, *d_A_indices, *d_B_indptr, *d_B_indices;
    float *d_A_data, *d_B_data;

    CUDA_CHECK(hipMalloc(&d_A_indptr, (A.rows + 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_A_indices, A.indices.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_A_data, A.data.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_B_indptr, (B.rows + 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_B_indices, B.indices.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_B_data, B.data.size() * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_A_indptr, A.indptr.data(), (A.rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_A_indices, A.indices.data(), A.indices.size() * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_A_data, A.data.data(), A.data.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B_indptr, B.indptr.data(), (B.rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B_indices, B.indices.data(), B.indices.size() * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B_data, B.data.data(), B.data.size() * sizeof(float), hipMemcpyHostToDevice));

    // --- 2. Allocate Persistent Workspaces ---
    int* d_workspace_markers;
    float* d_workspace_vals;
    size_t workspace_elements = static_cast<size_t>(A.rows) * B.cols;
    if (workspace_elements > 0) {
        CUDA_CHECK(hipMalloc(&d_workspace_markers, workspace_elements * sizeof(int)));
        CUDA_CHECK(hipMalloc(&d_workspace_vals, workspace_elements * sizeof(float)));
        CUDA_CHECK(hipMemset(d_workspace_markers, 0, workspace_elements * sizeof(int)));
    } else {
        d_workspace_markers = nullptr;
        d_workspace_vals = nullptr;
    }

    // --- 3. Symbolic Phase: Compute NNZ per row of C ---
    int* d_C_row_nnz;
    CUDA_CHECK(hipMalloc(&d_C_row_nnz, A.rows * sizeof(int)));

    const int warps_per_block = 8; // Tunable parameter
    dim3 threads(warpSize, warps_per_block, 1);
    dim3 blocks((A.rows + warps_per_block - 1) / warps_per_block, 1, 1);

    spgemm_symbolic_warp_per_row<<<blocks, threads>>>(A.rows, B.cols, d_A_indptr, d_A_indices, d_B_indptr, d_B_indices, d_C_row_nnz, d_workspace_markers);
    CUDA_CHECK(hipGetLastError());

    // --- 4. GPU-side Scan: Calculate C.indptr and total NNZ using CUB ---
    int *d_C_indptr;
    size_t c_indptr_size = (A.rows + 1) * sizeof(int);
    CUDA_CHECK(hipMalloc(&d_C_indptr, c_indptr_size));

    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    // First, get the size of the temporary storage CUB needs
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_C_row_nnz, d_C_indptr, A.rows);
    CUDA_CHECK(hipGetLastError());
    // Allocate the temporary storage
    CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));
    // Now, perform the scan
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_C_row_nnz, d_C_indptr, A.rows);
    CUDA_CHECK(hipGetLastError());

    int C_nnz = 0;
    if (A.rows > 0) {
        // To get the total NNZ, we need the last element of the indptr array plus the last element of the input array.
        int last_nnz_count;
        CUDA_CHECK(hipMemcpy(&last_nnz_count, d_C_row_nnz + A.rows - 1, sizeof(int), hipMemcpyDeviceToHost));
        int last_indptr_val;
        CUDA_CHECK(hipMemcpy(&last_indptr_val, d_C_indptr + A.rows - 1, sizeof(int), hipMemcpyDeviceToHost));
        C_nnz = last_indptr_val + last_nnz_count;
        // Copy the total NNZ to the last position of d_C_indptr on the device
        CUDA_CHECK(hipMemcpy(d_C_indptr + A.rows, &C_nnz, sizeof(int), hipMemcpyHostToDevice));
    }

    C.rows = A.rows;
    C.cols = B.cols;
    C.indptr.resize(A.rows + 1);
    if (C_nnz > 0) {
        C.indices.resize(C_nnz);
        C.data.resize(C_nnz);
    }

    CUDA_CHECK(hipMemcpy(C.indptr.data(), d_C_indptr, c_indptr_size, hipMemcpyDeviceToHost));

    // --- 5. Numeric Phase: Compute C.indices and C.data ---
    if (C_nnz > 0) {
        int* d_C_indices;
        float* d_C_data;
        CUDA_CHECK(hipMalloc(&d_C_indices, C_nnz * sizeof(int)));
        CUDA_CHECK(hipMalloc(&d_C_data, C_nnz * sizeof(float)));

        size_t shared_mem_size = warps_per_block * sizeof(int);
        spgemm_numeric_warp_per_row<<<blocks, threads, shared_mem_size>>>(A.rows, B.cols,
                                                               d_A_indptr, d_A_indices, d_A_data,
                                                               d_B_indptr, d_B_indices, d_B_data,
                                                               d_C_indptr, d_C_indices, d_C_data,
                                                               d_workspace_vals, d_workspace_markers);
        CUDA_CHECK(hipGetLastError());

        CUDA_CHECK(hipMemcpy(C.indices.data(), d_C_indices, C_nnz * sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(C.data.data(), d_C_data, C_nnz * sizeof(float), hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(d_C_indices));
        CUDA_CHECK(hipFree(d_C_data));
    }

    // --- 6. Cleanup ---
    CUDA_CHECK(hipFree(d_A_indptr));
    CUDA_CHECK(hipFree(d_A_indices));
    CUDA_CHECK(hipFree(d_A_data));
    CUDA_CHECK(hipFree(d_B_indptr));
    CUDA_CHECK(hipFree(d_B_indices));
    CUDA_CHECK(hipFree(d_B_data));
    CUDA_CHECK(hipFree(d_C_row_nnz));
    CUDA_CHECK(hipFree(d_C_indptr));
    CUDA_CHECK(hipFree(d_temp_storage));
    if (workspace_elements > 0) {
        CUDA_CHECK(hipFree(d_workspace_markers));
        CUDA_CHECK(hipFree(d_workspace_vals));
    }
}
