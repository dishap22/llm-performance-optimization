#include "hip/hip_runtime.h"
#include <vector>
#include <stdexcept>
#include <numeric>

#include <hip/hip_runtime.h>

// Use the correct relative path to the header file
// based on the project structure.
#include "../utils/csr_io.h"

// Helper macro for CUDA error checking.
#define CUDA_CHECK(err) { \
    hipError_t e = (err); \
    if (e != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(EXIT_FAILURE); \
    } \
}

/**
 * @brief A simple, single-threaded kernel to perform an exclusive scan (prefix sum) on the GPU.
 *
 * This kernel replaces the need for external libraries, avoiding compiler compatibility issues.
 * It runs on a single thread but avoids expensive device-to-host data transfers.
 * NOTE: For very large matrices (e.g., millions of rows), this serial scan can become a
 * bottleneck. A parallel scan implementation would be a further optimization.
 *
 * @param d_in      Device pointer to the input array (e.g., row NNZ counts).
 * @param d_out     Device pointer to the output array (the result of the scan).
 * @param n         The number of elements in the array.
 */
__global__ void exclusive_scan_kernel(const int* d_in, int* d_out, int n) {
    // This kernel is intended to be launched with a single thread.
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        int sum = 0;
        for (int i = 0; i < n; i++) {
            int temp = d_in[i];
            d_out[i] = sum;
            sum += temp;
        }
        // The last element of indptr is the total NNZ
        d_out[n] = sum;
    }
}


/**
 * @brief Optimized symbolic kernel using a "block-per-row" strategy.
 *
 * Each thread block calculates the number of non-zero elements for a single row of C.
 * This approach significantly improves GPU utilization by launching one block per row,
 * ensuring that many SMs are active. Threads within the block parallelize the computation.
 *
 * @param num_rows          The number of rows in matrix A (and C).
 * @param b_cols            The number of columns in matrix B (and C).
 * @param d_A_indptr        Device pointer to the indptr array of matrix A.
 * @param d_A_indices       Device pointer to the indices array of matrix A.
 * @param d_B_indptr        Device pointer to the indptr array of matrix B.
 * @param d_B_indices       Device pointer to the indices array of matrix B.
 * @param d_C_row_nnz       Device pointer to an array to store the NNZ count for each row of C.
 * @param d_workspace_markers Device pointer to an integer workspace of size (num_rows * b_cols).
 */
__global__ void spgemm_symbolic_block_per_row(const int num_rows, const int b_cols,
                                              const int* __restrict__ d_A_indptr, const int* __restrict__ d_A_indices,
                                              const int* __restrict__ d_B_indptr, const int* __restrict__ d_B_indices,
                                              int* d_C_row_nnz, int* d_workspace_markers) {
    // Each block computes one row of C.
    const int row = blockIdx.x;
    if (row >= num_rows) return;

    const int row_marker = row + 1;
    // Each block works on its own slice of the global workspace.
    int* markers = d_workspace_markers + static_cast<size_t>(row) * b_cols;

    const int start_A = d_A_indptr[row];
    const int end_A = d_A_indptr[row + 1];

    // Parallelize the loop over A's non-zeros across threads in the block.
    for (int i = start_A + threadIdx.x; i < end_A; i += blockDim.x) {
        const int col_A = d_A_indices[i];
        const int row_B = col_A;
        const int start_B = d_B_indptr[row_B];
        const int end_B = d_B_indptr[row_B + 1];

        for (int j = start_B; j < end_B; ++j) {
            const int col_B = d_B_indices[j];
            // A non-atomic write is sufficient. If multiple threads write the same
            // marker, the outcome is identical (benign race condition).
            markers[col_B] = row_marker;
        }
    }

    // Synchronize all threads in the block to ensure all markers are written before counting.
    __syncthreads();

    // Count the marked elements in parallel.
    int thread_nnz = 0;
    for (int j = threadIdx.x; j < b_cols; j += blockDim.x) {
        if (markers[j] == row_marker) {
            thread_nnz++;
        }
    }

    // Reduce the counts from all threads in the block using shared memory.
    extern __shared__ int s_nnz[];
    s_nnz[threadIdx.x] = thread_nnz;
    __syncthreads();

    // Perform parallel reduction in shared memory.
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            s_nnz[threadIdx.x] += s_nnz[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Thread 0 writes the final, total count for the row.
    if (threadIdx.x == 0) {
        d_C_row_nnz[row] = s_nnz[0];
    }
}


/**
 * @brief Corrected and optimized numeric kernel using a "block-per-row" strategy.
 *
 * This version fixes the race condition by using a zero-initialized workspace
 * and relying exclusively on atomicAdd for accumulation. This is thread-safe
 * and ensures numerical correctness.
 *
 * @param num_rows              The number of rows in matrix A (and C).
 * @param b_cols                The number of columns in matrix B (and C).
 * @param ...                   Device pointers for matrices A, B, and C.
 * @param d_workspace_vals      Device pointer to a float workspace, MUST be zero-initialized.
 * @param d_workspace_markers   Device pointer to an integer workspace from the symbolic phase.
 */
__global__ void spgemm_numeric_block_per_row(const int num_rows, const int b_cols,
                                             const int* __restrict__ d_A_indptr, const int* __restrict__ d_A_indices, const float* __restrict__ d_A_data,
                                             const int* __restrict__ d_B_indptr, const int* __restrict__ d_B_indices, const float* __restrict__ d_B_data,
                                             const int* __restrict__ d_C_indptr, int* d_C_indices, float* d_C_data,
                                             float* d_workspace_vals, int* d_workspace_markers) {
    // Each block computes one row of C.
    const int row = blockIdx.x;
    if (row >= num_rows) return;

    // The marker value set by the symbolic phase for this row.
    const int symbolic_marker = row + 1;

    // Each block works on its own slice of the global workspaces.
    float* acc_vals = d_workspace_vals + static_cast<size_t>(row) * b_cols;
    int* acc_markers = d_workspace_markers + static_cast<size_t>(row) * b_cols;

    const int start_A = d_A_indptr[row];
    const int end_A = d_A_indptr[row + 1];

    // Part 1: Accumulate products in parallel into the zero-initialized workspace.
    // This loop iterates over the non-zero elements of row A assigned to this thread.
    for (int i = start_A + threadIdx.x; i < end_A; i += blockDim.x) {
        const int col_A = d_A_indices[i];
        const float val_A = d_A_data[i];
        const int row_B = col_A;

        const int start_B = d_B_indptr[row_B];
        const int end_B = d_B_indptr[row_B + 1];

        // For each non-zero in A, iterate over the corresponding row in B.
        for (int j = start_B; j < end_B; ++j) {
            const int col_B = d_B_indices[j];
            const float val_B = d_B_data[j];

            // Atomically add the product to the accumulator for the corresponding output column.
            // Since the workspace is pre-initialized to zero, we can just use atomicAdd
            // for all updates, which is race-free.
            atomicAdd(&acc_vals[col_B], val_A * val_B);
        }
    }

    // Synchronize all threads in the block to ensure all atomic additions are globally visible
    // before the compaction phase begins.
    __syncthreads();

    // Part 2: Write the compacted results from the workspace to global memory.
    // This is done serially by a single thread (thread 0) in the block.
    if (threadIdx.x == 0) {
        const int C_row_start = d_C_indptr[row];
        int C_nnz_written = 0;
        // Scan through all possible columns for this row's slice of the workspace.
        for (int j = 0; j < b_cols; ++j) {
            // Use the marker from the symbolic phase to identify the non-zero columns.
            if (acc_markers[j] == symbolic_marker) {
                d_C_indices[C_row_start + C_nnz_written] = j;
                d_C_data[C_row_start + C_nnz_written] = acc_vals[j];
                C_nnz_written++;
            }
        }
    }
}


/**
 * @brief Performs sparse matrix-matrix multiplication (SpGEMM) on the GPU.
 *
 * This version is fully self-contained and optimized for performance by using a
 * "block-per-row" strategy, which ensures high GPU occupancy and numerical correctness.
 *
 * @param A The first input sparse matrix in CSR format.
 * @param B The second input sparse matrix in CSR format.
 * @param C The output sparse matrix in CSR format.
 */
void spgemm_gpu(const CSRMatrix& A, const CSRMatrix& B, CSRMatrix& C) {
    if (A.cols != B.rows) {
        throw std::runtime_error("Incompatible matrix dimensions for SpGEMM (A.cols must equal B.rows).");
    }

    // --- 1. Device Memory Allocation & H2D Transfer for Inputs ---
    int *d_A_indptr, *d_A_indices, *d_B_indptr, *d_B_indices;
    float *d_A_data, *d_B_data;

    CUDA_CHECK(hipMalloc(&d_A_indptr, (A.rows + 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_A_indices, A.indices.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_A_data, A.data.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_B_indptr, (B.rows + 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_B_indices, B.indices.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_B_data, B.data.size() * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_A_indptr, A.indptr.data(), (A.rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_A_indices, A.indices.data(), A.indices.size() * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_A_data, A.data.data(), A.data.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B_indptr, B.indptr.data(), (B.rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B_indices, B.indices.data(), B.indices.size() * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B_data, B.data.data(), B.data.size() * sizeof(float), hipMemcpyHostToDevice));

    // --- 2. Allocate and Initialize Persistent Workspaces ---
    int* d_workspace_markers;
    float* d_workspace_vals;
    size_t workspace_elements = static_cast<size_t>(A.rows) * B.cols;
    if (workspace_elements > 0) {
        CUDA_CHECK(hipMalloc(&d_workspace_markers, workspace_elements * sizeof(int)));
        CUDA_CHECK(hipMalloc(&d_workspace_vals, workspace_elements * sizeof(float)));
        // The marker-based algorithm relies on an initial zeroed state.
        CUDA_CHECK(hipMemset(d_workspace_markers, 0, workspace_elements * sizeof(int)));
        // FIX: The value workspace must also be zero-initialized for the atomicAdd approach.
        CUDA_CHECK(hipMemset(d_workspace_vals, 0, workspace_elements * sizeof(float)));
    } else {
        d_workspace_markers = nullptr;
        d_workspace_vals = nullptr;
    }

    // --- 3. Symbolic Phase: Compute NNZ per row of C ---
    int* d_C_row_nnz;
    CUDA_CHECK(hipMalloc(&d_C_row_nnz, A.rows * sizeof(int)));

    // Set launch configuration for "block-per-row" strategy.
    int threads_per_block = 256;
    int blocks_per_grid = A.rows; // Launch one block for each row of A
    size_t shared_mem_size = threads_per_block * sizeof(int); // For reduction in symbolic kernel

    if (blocks_per_grid > 0) {
        spgemm_symbolic_block_per_row<<<blocks_per_grid, threads_per_block, shared_mem_size>>>(
            A.rows, B.cols, d_A_indptr, d_A_indices, d_B_indptr, d_B_indices, d_C_row_nnz, d_workspace_markers);
        CUDA_CHECK(hipGetLastError());
    }

    // --- 4. GPU-side Scan: Calculate C.indptr and total NNZ ---
    int *d_C_indptr;
    size_t c_indptr_size = (A.rows + 1) * sizeof(int);
    CUDA_CHECK(hipMalloc(&d_C_indptr, c_indptr_size));

    if (A.rows > 0) {
        exclusive_scan_kernel<<<1, 1>>>(d_C_row_nnz, d_C_indptr, A.rows);
        CUDA_CHECK(hipGetLastError());
    }

    int C_nnz = 0;
    if (A.rows > 0) {
        CUDA_CHECK(hipMemcpy(&C_nnz, d_C_indptr + A.rows, sizeof(int), hipMemcpyDeviceToHost));
    }

    C.rows = A.rows;
    C.cols = B.cols;
    C.indptr.resize(A.rows + 1);
    C.indices.resize(C_nnz);
    C.data.resize(C_nnz);

    if (A.rows > 0) {
        CUDA_CHECK(hipMemcpy(C.indptr.data(), d_C_indptr, c_indptr_size, hipMemcpyDeviceToHost));
    }

    // --- 5. Numeric Phase: Compute C.indices and C.data ---
    if (C_nnz > 0) {
        int* d_C_indices;
        float* d_C_data;
        CUDA_CHECK(hipMalloc(&d_C_indices, C_nnz * sizeof(int)));
        CUDA_CHECK(hipMalloc(&d_C_data, C_nnz * sizeof(float)));

        if (blocks_per_grid > 0) {
            spgemm_numeric_block_per_row<<<blocks_per_grid, threads_per_block>>>(
                A.rows, B.cols,
                d_A_indptr, d_A_indices, d_A_data,
                d_B_indptr, d_B_indices, d_B_data,
                d_C_indptr, d_C_indices, d_C_data,
                d_workspace_vals, d_workspace_markers);
            CUDA_CHECK(hipGetLastError());
        }

        CUDA_CHECK(hipMemcpy(C.indices.data(), d_C_indices, C_nnz * sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(C.data.data(), d_C_data, C_nnz * sizeof(float), hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(d_C_indices));
        CUDA_CHECK(hipFree(d_C_data));
    }

    // --- 6. Cleanup ---
    CUDA_CHECK(hipFree(d_A_indptr));
    CUDA_CHECK(hipFree(d_A_indices));
    CUDA_CHECK(hipFree(d_A_data));
    CUDA_CHECK(hipFree(d_B_indptr));
    CUDA_CHECK(hipFree(d_B_indices));
    CUDA_CHECK(hipFree(d_B_data));
    CUDA_CHECK(hipFree(d_C_row_nnz));
    CUDA_CHECK(hipFree(d_C_indptr));
    if (workspace_elements > 0) {
        CUDA_CHECK(hipFree(d_workspace_markers));
        CUDA_CHECK(hipFree(d_workspace_vals));
    }
}
