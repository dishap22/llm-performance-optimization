#include "hip/hip_runtime.h"
#include <vector>
#include <stdexcept>
#include <numeric>

#include <hip/hip_runtime.h>

// Use the correct relative path to the header file
// based on the project structure.
#include "../utils/csr_io.h"

// Helper macro for CUDA error checking.
#define CUDA_CHECK(err) { \
    hipError_t e = (err); \
    if (e != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(EXIT_FAILURE); \
    } \
}

/**
 * @brief A simple, single-threaded kernel to perform an exclusive scan (prefix sum) on the GPU.
 *
 * This kernel replaces the need for external libraries, avoiding compiler compatibility issues.
 * It runs on a single thread but avoids expensive device-to-host data transfers.
 *
 * @param d_in      Device pointer to the input array (e.g., row NNZ counts).
 * @param d_out     Device pointer to the output array (the result of the scan).
 * @param n         The number of elements in the array.
 */
__global__ void exclusive_scan_kernel(const int* d_in, int* d_out, int n) {
    // This kernel is intended to be launched with a single thread.
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        int sum = 0;
        for (int i = 0; i < n; i++) {
            int temp = d_in[i];
            d_out[i] = sum;
            sum += temp;
        }
        // The last element of indptr is the total NNZ
        d_out[n] = sum;
    }
}


/**
 * @brief Optimized symbolic kernel to calculate the number of non-zero elements per row of C.
 *
 * This version uses a "marker" algorithm to avoid expensive hipMemset operations on the workspace.
 *
 * @param num_rows          The number of rows in matrix A (and C).
 * @param b_cols            The number of columns in matrix B (and C).
 * @param d_A_indptr        Device pointer to the indptr array of matrix A.
 * @param d_A_indices       Device pointer to the indices array of matrix A.
 * @param d_B_indptr        Device pointer to the indptr array of matrix B.
 * @param d_B_indices       Device pointer to the indices array of matrix B.
 * @param d_C_row_nnz       Device pointer to an array to store the NNZ count for each row of C.
 * @param d_workspace_markers Device pointer to an integer workspace of size (num_rows * b_cols).
 */
__global__ void spgemm_symbolic_optimized(const int num_rows, const int b_cols,
                                          const int* __restrict__ d_A_indptr, const int* __restrict__ d_A_indices,
                                          const int* __restrict__ d_B_indptr, const int* __restrict__ d_B_indices,
                                          int* d_C_row_nnz, int* d_workspace_markers) {
    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= num_rows) return;

    const int row_marker = row + 1;
    int* markers = d_workspace_markers + static_cast<size_t>(row) * b_cols;
    int nnz = 0;
    const int start_A = d_A_indptr[row];
    const int end_A = d_A_indptr[row + 1];

    for (int i = start_A; i < end_A; ++i) {
        const int col_A = d_A_indices[i];
        const int row_B = col_A;
        const int start_B = d_B_indptr[row_B];
        const int end_B = d_B_indptr[row_B + 1];

        for (int j = start_B; j < end_B; ++j) {
            const int col_B = d_B_indices[j];
            if (markers[col_B] != row_marker) {
                markers[col_B] = row_marker;
                nnz++;
            }
        }
    }
    d_C_row_nnz[row] = nnz;
}


/**
 * @brief Optimized numeric kernel to compute the values and column indices of the output matrix C.
 *
 * This version uses the marker algorithm to avoid expensive workspace clearing. It reverts
 * to a full scan over the columns for the final compaction step to guarantee correctness
 * for dense output rows, fixing a bug where a fixed-size local array would overflow.
 *
 * @param num_rows              The number of rows in matrix A (and C).
 * @param b_cols                The number of columns in matrix B (and C).
 * @param ...                   Device pointers for matrices A, B, and C.
 * @param d_workspace_vals      Device pointer to a float workspace for accumulating values.
 * @param d_workspace_markers   Device pointer to an integer workspace for tracking active accumulators.
 */
__global__ void spgemm_numeric_optimized(const int num_rows, const int b_cols,
                                         const int* __restrict__ d_A_indptr, const int* __restrict__ d_A_indices, const float* __restrict__ d_A_data,
                                         const int* __restrict__ d_B_indptr, const int* __restrict__ d_B_indices, const float* __restrict__ d_B_data,
                                         const int* __restrict__ d_C_indptr, int* d_C_indices, float* d_C_data,
                                         float* d_workspace_vals, int* d_workspace_markers) {
    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= num_rows) return;

    // Use a different range of markers for the numeric phase to distinguish
    // it from the symbolic phase.
    const int numeric_row_marker = row + 1 + num_rows;

    float* acc_vals = d_workspace_vals + static_cast<size_t>(row) * b_cols;
    int* acc_markers = d_workspace_markers + static_cast<size_t>(row) * b_cols;

    const int start_A = d_A_indptr[row];
    const int end_A = d_A_indptr[row + 1];

    // Part 1: Accumulate products for the current row into the workspace.
    for (int i = start_A; i < end_A; ++i) {
        const int col_A = d_A_indices[i];
        const float val_A = d_A_data[i];
        const int row_B = col_A;

        const int start_B = d_B_indptr[row_B];
        const int end_B = d_B_indptr[row_B + 1];

        for (int j = start_B; j < end_B; ++j) {
            const int col_B = d_B_indices[j];
            const float val_B = d_B_data[j];

            if (acc_markers[col_B] != numeric_row_marker) {
                acc_markers[col_B] = numeric_row_marker;
                acc_vals[col_B] = val_A * val_B;
            } else {
                acc_vals[col_B] += val_A * val_B;
            }
        }
    }

    // Part 2: Write the compacted results from the workspace to global memory.
    // This part is reverted to the safer, albeit slower, full scan to fix the
    // bug caused by the fixed-size local temp array overflowing. This guarantees
    // correctness for any output sparsity.
    const int C_row_start = d_C_indptr[row];
    int C_nnz_written = 0;
    for (int j = 0; j < b_cols; ++j) {
        if (acc_markers[j] == numeric_row_marker) {
            d_C_indices[C_row_start + C_nnz_written] = j;
            d_C_data[C_row_start + C_nnz_written] = acc_vals[j];
            C_nnz_written++;
        }
    }
}


/**
 * @brief Performs sparse matrix-matrix multiplication (SpGEMM) on the GPU.
 *
 * This version is fully self-contained and optimized for correctness and performance,
 * avoiding external libraries and unsafe assumptions about data sparsity.
 *
 * @param A The first input sparse matrix in CSR format.
 * @param B The second input sparse matrix in CSR format.
 * @param C The output sparse matrix in CSR format.
 */
void spgemm_gpu(const CSRMatrix& A, const CSRMatrix& B, CSRMatrix& C) {
    if (A.cols != B.rows) {
        throw std::runtime_error("Incompatible matrix dimensions for SpGEMM (A.cols must equal B.rows).");
    }

    // --- 1. Device Memory Allocation & H2D Transfer for Inputs ---
    int *d_A_indptr, *d_A_indices, *d_B_indptr, *d_B_indices;
    float *d_A_data, *d_B_data;

    CUDA_CHECK(hipMalloc(&d_A_indptr, (A.rows + 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_A_indices, A.indices.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_A_data, A.data.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_B_indptr, (B.rows + 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_B_indices, B.indices.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_B_data, B.data.size() * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_A_indptr, A.indptr.data(), (A.rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_A_indices, A.indices.data(), A.indices.size() * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_A_data, A.data.data(), A.data.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B_indptr, B.indptr.data(), (B.rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B_indices, B.indices.data(), B.indices.size() * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B_data, B.data.data(), B.data.size() * sizeof(float), hipMemcpyHostToDevice));

    // --- 2. Allocate Persistent Workspaces ---
    int* d_workspace_markers;
    float* d_workspace_vals;
    size_t workspace_elements = static_cast<size_t>(A.rows) * B.cols;
    if (workspace_elements > 0) {
        CUDA_CHECK(hipMalloc(&d_workspace_markers, workspace_elements * sizeof(int)));
        CUDA_CHECK(hipMalloc(&d_workspace_vals, workspace_elements * sizeof(float)));
        CUDA_CHECK(hipMemset(d_workspace_markers, 0, workspace_elements * sizeof(int)));
    } else {
        d_workspace_markers = nullptr;
        d_workspace_vals = nullptr;
    }

    // --- 3. Symbolic Phase: Compute NNZ per row of C ---
    int* d_C_row_nnz;
    CUDA_CHECK(hipMalloc(&d_C_row_nnz, A.rows * sizeof(int)));

    int threads_per_block = 256;
    int blocks_per_grid = (A.rows + threads_per_block - 1) / threads_per_block;

    spgemm_symbolic_optimized<<<blocks_per_grid, threads_per_block>>>(A.rows, B.cols, d_A_indptr, d_A_indices, d_B_indptr, d_B_indices, d_C_row_nnz, d_workspace_markers);
    CUDA_CHECK(hipGetLastError());

    // --- 4. GPU-side Scan: Calculate C.indptr and total NNZ ---
    int *d_C_indptr;
    size_t c_indptr_size = (A.rows + 1) * sizeof(int);
    CUDA_CHECK(hipMalloc(&d_C_indptr, c_indptr_size));

    // Call the custom kernel to perform exclusive scan on the GPU.
    if (A.rows > 0) {
        exclusive_scan_kernel<<<1, 1>>>(d_C_row_nnz, d_C_indptr, A.rows);
        CUDA_CHECK(hipGetLastError());
    }

    int C_nnz = 0;
    if (A.rows > 0) {
        CUDA_CHECK(hipMemcpy(&C_nnz, d_C_indptr + A.rows, sizeof(int), hipMemcpyDeviceToHost));
    }

    C.rows = A.rows;
    C.cols = B.cols;
    C.indptr.resize(A.rows + 1);
    C.indices.resize(C_nnz);
    C.data.resize(C_nnz);

    CUDA_CHECK(hipMemcpy(C.indptr.data(), d_C_indptr, c_indptr_size, hipMemcpyDeviceToHost));

    // --- 5. Numeric Phase: Compute C.indices and C.data ---
    if (C_nnz > 0) {
        int* d_C_indices;
        float* d_C_data;
        CUDA_CHECK(hipMalloc(&d_C_indices, C_nnz * sizeof(int)));
        CUDA_CHECK(hipMalloc(&d_C_data, C_nnz * sizeof(float)));

        spgemm_numeric_optimized<<<blocks_per_grid, threads_per_block>>>(A.rows, B.cols,
                                                               d_A_indptr, d_A_indices, d_A_data,
                                                               d_B_indptr, d_B_indices, d_B_data,
                                                               d_C_indptr, d_C_indices, d_C_data,
                                                               d_workspace_vals, d_workspace_markers);
        CUDA_CHECK(hipGetLastError());

        CUDA_CHECK(hipMemcpy(C.indices.data(), d_C_indices, C_nnz * sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(C.data.data(), d_C_data, C_nnz * sizeof(float), hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(d_C_indices));
        CUDA_CHECK(hipFree(d_C_data));
    }

    // --- 6. Cleanup ---
    CUDA_CHECK(hipFree(d_A_indptr));
    CUDA_CHECK(hipFree(d_A_indices));
    CUDA_CHECK(hipFree(d_A_data));
    CUDA_CHECK(hipFree(d_B_indptr));
    CUDA_CHECK(hipFree(d_B_indices));
    CUDA_CHECK(hipFree(d_B_data));
    CUDA_CHECK(hipFree(d_C_row_nnz));
    CUDA_CHECK(hipFree(d_C_indptr));
    if (workspace_elements > 0) {
        CUDA_CHECK(hipFree(d_workspace_markers));
        CUDA_CHECK(hipFree(d_workspace_vals));
    }
}
