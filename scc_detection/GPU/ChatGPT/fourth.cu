#include "hip/hip_runtime.h"
// CUDA version of the SCC Solver based on Method 2 (Trim, Trim2, WCC, FWBW)
// Graph is read from a SNAP-style .txt file

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <sstream>
#include <unordered_map>
#include <set>
#include <queue>

#define THREADS_PER_BLOCK 512

struct Graph {
    int num_nodes;
    int num_edges;
    std::vector<int> row_offsets;
    std::vector<int> col_indices;
};

__global__ void compute_degrees(int* row_offsets, int* col_indices, int* in_deg, int* out_deg, int* marks, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= N || marks[tid]) return;
    out_deg[tid] = row_offsets[tid + 1] - row_offsets[tid];
    for (int i = row_offsets[tid]; i < row_offsets[tid + 1]; ++i) {
        int v = col_indices[i];
        if (!marks[v]) atomicAdd(&in_deg[v], 1);
    }
}

__global__ void trim_kernel(int* in_deg, int* out_deg, int* marks, int* colors, int N, bool* changed) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= N || marks[tid]) return;
    if (in_deg[tid] == 0 || out_deg[tid] == 0) {
        marks[tid] = 1;
        colors[tid] = -1;
        *changed = true;
    }
}

__global__ void wcc_label_propagation(int* row_offsets, int* col_indices, int* labels, int* marks, bool* changed, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= N || marks[tid]) return;
    int my_label = labels[tid];
    for (int i = row_offsets[tid]; i < row_offsets[tid + 1]; ++i) {
        int nbr = col_indices[i];
        if (marks[nbr]) continue;
        if (labels[nbr] > my_label) {
            atomicMin(&labels[nbr], my_label);
            *changed = true;
        } else if (labels[nbr] < my_label) {
            atomicMin(&labels[tid], labels[nbr]);
            *changed = true;
        }
    }
}

__global__ void bfs_kernel(int* row_offsets, int* col_indices, int* active, int* visited, int* marks, int* labels, int label, int N, bool* changed, bool forward) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= N || !active[tid] || visited[tid]) return;
    visited[tid] = 1;
    active[tid] = 0;

    if (forward) {
        for (int i = row_offsets[tid]; i < row_offsets[tid + 1]; ++i) {
            int nbr = col_indices[i];
            if (!marks[nbr] && labels[nbr] == label && !visited[nbr]) {
                active[nbr] = 1;
                *changed = true;
            }
        }
    } else {
        for (int u = 0; u < N; ++u) {
            for (int i = row_offsets[u]; i < row_offsets[u + 1]; ++i) {
                if (col_indices[i] == tid && !marks[u] && labels[u] == label && !visited[u]) {
                    active[u] = 1;
                    *changed = true;
                }
            }
        }
    }
}

Graph load_graph_from_file(const std::string& filename) {
    std::ifstream infile(filename);
    std::vector<std::pair<int, int>> edges;
    int max_node = 0;
    std::string line;
    while (std::getline(infile, line)) {
        if (line.empty() || line[0] == '#') continue;
        std::istringstream iss(line);
        int u, v;
        if (iss >> u >> v) {
            edges.emplace_back(u, v);
            max_node = std::max({max_node, u, v});
        }
    }
    int n = max_node + 1;
    std::vector<int> out_deg(n, 0);
    for (auto& e : edges) out_deg[e.first]++;
    std::vector<int> row_offsets(n + 1, 0);
    for (int i = 0; i < n; ++i) row_offsets[i + 1] = row_offsets[i] + out_deg[i];
    std::vector<int> col_indices(edges.size());
    std::vector<int> counter(n, 0);
    for (auto& e : edges) {
        int idx = row_offsets[e.first] + counter[e.first]++;
        col_indices[idx] = e.second;
    }
    return Graph{n, static_cast<int>(edges.size()), row_offsets, col_indices};
}

void SCC_Method2_CUDA(Graph& G) {
    int N = G.num_nodes;

    int *d_row_offsets, *d_col_indices, *d_in_deg, *d_out_deg, *d_marks, *d_colors, *d_labels;
    bool *d_changed;
    hipMalloc(&d_row_offsets, sizeof(int) * G.row_offsets.size());
    hipMalloc(&d_col_indices, sizeof(int) * G.col_indices.size());
    hipMalloc(&d_in_deg, sizeof(int) * N);
    hipMalloc(&d_out_deg, sizeof(int) * N);
    hipMalloc(&d_marks, sizeof(int) * N);
    hipMalloc(&d_colors, sizeof(int) * N);
    hipMalloc(&d_labels, sizeof(int) * N);
    hipMalloc(&d_changed, sizeof(bool));

    hipMemcpy(d_row_offsets, G.row_offsets.data(), sizeof(int) * G.row_offsets.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_col_indices, G.col_indices.data(), sizeof(int) * G.col_indices.size(), hipMemcpyHostToDevice);
    hipMemset(d_marks, 0, sizeof(int) * N);
    hipMemset(d_colors, 0, sizeof(int) * N);

    thrust::device_vector<int> d_labels_vec(N);
    thrust::sequence(d_labels_vec.begin(), d_labels_vec.end());
    hipMemcpy(d_labels, thrust::raw_pointer_cast(d_labels_vec.data()), sizeof(int) * N, hipMemcpyDeviceToDevice);

    bool h_changed = true;
    while (h_changed) {
        h_changed = false;
        hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice);
        hipMemset(d_in_deg, 0, sizeof(int) * N);
        hipMemset(d_out_deg, 0, sizeof(int) * N);
        compute_degrees<<<(N + 511)/512, 512>>>(d_row_offsets, d_col_indices, d_in_deg, d_out_deg, d_marks, N);
        trim_kernel<<<(N + 511)/512, 512>>>(d_in_deg, d_out_deg, d_marks, d_colors, N, d_changed);
        hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost);
    }

    // WCC
    h_changed = true;
    while (h_changed) {
        h_changed = false;
        hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice);
        wcc_label_propagation<<<(N + 511)/512, 512>>>(d_row_offsets, d_col_indices, d_labels, d_marks, d_changed, N);
        hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost);
    }

    thrust::host_vector<int> h_labels(N);
    hipMemcpy(h_labels.data(), d_labels, sizeof(int) * N, hipMemcpyDeviceToHost);

    std::unordered_map<int, std::vector<int>> label_groups;
    for (int i = 0; i < N; ++i) label_groups[h_labels[i]].push_back(i);

    thrust::device_vector<int> d_active(N);
    thrust::device_vector<int> d_fw(N);
    thrust::device_vector<int> d_bw(N);
    std::set<std::set<int>> scc_set;

    for (const auto& [label, group] : label_groups) {
        thrust::fill(d_active.begin(), d_active.end(), 0);
        thrust::fill(d_fw.begin(), d_fw.end(), 0);
        thrust::fill(d_bw.begin(), d_bw.end(), 0);

        d_active[group[0]] = 1;
        h_changed = true;
        while (h_changed) {
            h_changed = false;
            hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice);
            bfs_kernel<<<(N + 511)/512, 512>>>(d_row_offsets, d_col_indices,
                thrust::raw_pointer_cast(d_active.data()),
                thrust::raw_pointer_cast(d_fw.data()),
                d_marks, d_labels, label, N, d_changed, true);
            hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost);
        }

        d_active[group[0]] = 1;
        h_changed = true;
        while (h_changed) {
            h_changed = false;
            hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice);
            bfs_kernel<<<(N + 511)/512, 512>>>(d_row_offsets, d_col_indices,
                thrust::raw_pointer_cast(d_active.data()),
                thrust::raw_pointer_cast(d_bw.data()),
                d_marks, d_labels, label, N, d_changed, false);
            hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost);
        }

        thrust::host_vector<int> h_fw = d_fw;
        thrust::host_vector<int> h_bw = d_bw;
        std::set<int> scc;
        for (int i = 0; i < N; ++i) if (h_fw[i] && h_bw[i]) scc.insert(i);
        if (!scc.empty()) scc_set.insert(scc);
    }

    std::cout << "\nTotal SCCs found: " << scc_set.size() << "\n";


    hipFree(d_row_offsets);
    hipFree(d_col_indices);
    hipFree(d_in_deg);
    hipFree(d_out_deg);
    hipFree(d_marks);
    hipFree(d_colors);
    hipFree(d_labels);
    hipFree(d_changed);
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <graph_file.txt>\n";
        return 1;
    }
    std::string filename = argv[1];
    Graph G = load_graph_from_file(filename);
    std::cout << "Graph loaded with " << G.num_nodes << " nodes and " << G.num_edges << " edges.\n";
    SCC_Method2_CUDA(G);
    return 0;
}
